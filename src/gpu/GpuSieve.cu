#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include "GpuSieve.hpp"
#include <iostream>
#include <vector>
#include <cassert>

// Maximum stride for GPU processing
const uint64_t MAX_STRIDE = 256000000;
// Number of threads per block (adjustable based on GPU)
int BLOCK_SIZE = 256;

// Constructor: Initialize GpuSieve object with no allocated memory on the host
GpuSieve::GpuSieve() : maxLimit(0), isPrimeListHost(nullptr) {}

// Destructor: Free host memory if allocated
GpuSieve::~GpuSieve() {
    // Free host memory
    if (isPrimeListHost) {
        free(isPrimeListHost);
        isPrimeListHost = nullptr;
    }
}

// Set the maximum limit for prime computation
void GpuSieve::setMaxLimit(unsigned int maxLimit) {
    this->maxLimit = maxLimit;
}

// Main function to compute primes up to maxLimit
void GpuSieve::computePrimes() {
    // Prepare initial list of small primes on CPU using a simple sieve
    std::vector<uint64_t> prepedPrimes = sieveCpuPrep(std::sqrt(maxLimit));

    // Launch GPU sieve to find all primes up to maxLimit
    gpuSieve(maxLimit, prepedPrimes);
}

// Retrieve the list of primes found
const std::vector<unsigned int>& GpuSieve::getPrimes() const {
    return primes;
}

// Collect primes from the result list on the host
void GpuSieve::collectPrimes() {
    // Clear any existing primes in the list
    primes.clear();
    // Traverse isPrimeListHost to collect primes up to maxLimit
    for (uint64_t i = 2; i <= maxLimit; i++) {
        if (isPrimeListHost[i]) {
            primes.push_back(i);
        }
    }
}

// Prepare a list of initial small primes on the CPU (sieve of Eratosthenes)
std::vector<uint64_t> GpuSieve::sieveCpuPrep(uint64_t maxNumber) {
    std::vector<bool> isPrimeList(maxNumber, true);
    std::vector<uint64_t> prepedPrimes;

    // Basic sieve algorithm to mark non-primes
    for (uint64_t i = 2; i < maxNumber; i++) {
        if (isPrimeList[i]) {
            prepedPrimes.push_back(i);
            // Mark multiples of i as non-prime
            for (uint64_t j = 2; i * j < maxNumber; j++) {
                isPrimeList[i * j] = false;
            }
        }
    }
    return prepedPrimes;
}

// CUDA kernel to perform the sieve on the GPU
__global__ void gpuSieveKernel(uint64_t maxNumber, bool* isPrimeList, uint64_t* prepedPrimes, uint64_t sizeOfPrepedPrimes) {
    uint64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    // Iterate over each position in the range for this thread
    for (uint64_t i = threadIndex; i <= maxNumber; i += stride) {
        if (i < 2) continue;
        // Mark multiples of precomputed primes as non-prime
        for (uint64_t j = 0; j < sizeOfPrepedPrimes; j++) {
            uint64_t currentNumber = prepedPrimes[j] * i;
            if (currentNumber > maxNumber) break;
            isPrimeList[currentNumber] = false;
        }
    }
}

// Function to launch GPU sieve with prepared primes
void GpuSieve::gpuSieve(uint64_t maxNumber, const std::vector<uint64_t>& prepedPrimes) {
    bool* isPrimeListDevice = nullptr;
    uint64_t* prepedPrimesDevice = nullptr;

    // Allocate and initialize memory on host and device
    isPrimeListHost = (bool*)malloc(maxNumber * sizeof(bool));
    std::memset(isPrimeListHost, true, maxNumber * sizeof(bool));
    hipMalloc(&isPrimeListDevice, maxNumber * sizeof(bool));
    hipMemcpy(isPrimeListDevice, isPrimeListHost, maxNumber * sizeof(bool), hipMemcpyHostToDevice);

    // Prepare and copy the list of initial primes to device
    uint64_t prepedPrimesSize = prepedPrimes.size();
    uint64_t* prepedPrimesHost = (uint64_t*)malloc(prepedPrimesSize * sizeof(uint64_t));
    memcpy(prepedPrimesHost, prepedPrimes.data(), prepedPrimesSize * sizeof(uint64_t));
    hipMalloc(&prepedPrimesDevice, prepedPrimesSize * sizeof(uint64_t));
    hipMemcpy(prepedPrimesDevice, prepedPrimesHost, prepedPrimesSize * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Configure blocks and threads for kernel launch
    uint64_t numberOfBlocks = (maxNumber + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if (numberOfBlocks * BLOCK_SIZE > MAX_STRIDE) {
        numberOfBlocks = MAX_STRIDE / BLOCK_SIZE;
    }

    // Launch the kernel
    gpuSieveKernel << <numberOfBlocks, BLOCK_SIZE >> > (maxNumber, isPrimeListDevice, prepedPrimesDevice, prepedPrimesSize);
    hipDeviceSynchronize();

    // Copy results back to the host
    hipMemcpy(isPrimeListHost, isPrimeListDevice, maxNumber * sizeof(bool), hipMemcpyDeviceToHost);

    // Clean up memory
    free(prepedPrimesHost);
    hipFree(isPrimeListDevice);
    hipFree(prepedPrimesDevice);
}
