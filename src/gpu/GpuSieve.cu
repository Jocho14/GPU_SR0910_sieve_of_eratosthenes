#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include "GpuSieve.hpp"
#include <iostream>
#include <vector>
#include <cassert>

// Sta�e i zmienne globalne dla GPU
const uint64_t MAX_STRIDE = 256000000;
int BLOCK_SIZE = 256;

GpuSieve::GpuSieve() : maxLimit(0), isPrimeListHost(nullptr) {}

GpuSieve::~GpuSieve() {
    // Zwolnienie pami�ci hosta
    if (isPrimeListHost) {
        free(isPrimeListHost);
        isPrimeListHost = nullptr;
    }
}

void GpuSieve::setMaxLimit(unsigned int maxLimit) {
    this->maxLimit = maxLimit;
}

void GpuSieve::computePrimes() {
    // Przygotowanie wst�pnych liczb pierwszych na CPU
    std::vector<uint64_t> prepedPrimes = sieveCpuPrep(std::sqrt(maxLimit));

    // Wywo�anie sita na GPU
    gpuSieve(maxLimit, prepedPrimes);
}

const std::vector<unsigned int>& GpuSieve::getPrimes() const {
    return primes;
}

void GpuSieve::collectPrimes() {
    // Zbieranie liczb pierwszych z listy wynik�w na ho�cie
    primes.clear();
    for (uint64_t i = 2; i <= maxLimit; i++) {
        if (isPrimeListHost[i]) {
            primes.push_back(i);
        }
    }
}

std::vector<uint64_t> GpuSieve::sieveCpuPrep(uint64_t maxNumber) {
    std::vector<bool> isPrimeList(maxNumber, true);
    std::vector<uint64_t> prepedPrimes;

    for (uint64_t i = 2; i < maxNumber; i++) {
        if (isPrimeList[i]) {
            prepedPrimes.push_back(i);
            for (uint64_t j = 2; i * j < maxNumber; j++) {
                isPrimeList[i * j] = false;
            }
        }
    }
    return prepedPrimes;
}

__global__ void gpuSieveKernel(uint64_t maxNumber, bool* isPrimeList, uint64_t* prepedPrimes, uint64_t sizeOfPrepedPrimes) {
    uint64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = threadIndex; i <= maxNumber; i += stride) {
        if (i < 2) continue;
        for (uint64_t j = 0; j < sizeOfPrepedPrimes; j++) {
            uint64_t currentNumber = prepedPrimes[j] * i;
            if (currentNumber > maxNumber) break;
            isPrimeList[currentNumber] = false;
        }
    }
}

void GpuSieve::gpuSieve(uint64_t maxNumber, const std::vector<uint64_t>& prepedPrimes) {
    bool* isPrimeListDevice = nullptr;
    uint64_t* prepedPrimesDevice = nullptr;

    // Alokacja i inicjalizacja pami�ci na ho�cie i urz�dzeniu
    isPrimeListHost = (bool*)malloc(maxNumber * sizeof(bool));
    std::memset(isPrimeListHost, true, maxNumber * sizeof(bool));
    hipMalloc(&isPrimeListDevice, maxNumber * sizeof(bool));
    hipMemcpy(isPrimeListDevice, isPrimeListHost, maxNumber * sizeof(bool), hipMemcpyHostToDevice);

    uint64_t prepedPrimesSize = prepedPrimes.size();
    uint64_t* prepedPrimesHost = (uint64_t*)malloc(prepedPrimesSize * sizeof(uint64_t));
    memcpy(prepedPrimesHost, prepedPrimes.data(), prepedPrimesSize * sizeof(uint64_t));
    hipMalloc(&prepedPrimesDevice, prepedPrimesSize * sizeof(uint64_t));
    hipMemcpy(prepedPrimesDevice, prepedPrimesHost, prepedPrimesSize * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Konfiguracja blok�w i w�tk�w dla kernela
    uint64_t numberOfBlocks = (maxNumber + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if (numberOfBlocks * BLOCK_SIZE > MAX_STRIDE) {
        numberOfBlocks = MAX_STRIDE / BLOCK_SIZE;
    }

    // Uruchomienie kernela
    gpuSieveKernel << <numberOfBlocks, BLOCK_SIZE >> > (maxNumber, isPrimeListDevice, prepedPrimesDevice, prepedPrimesSize);
    hipDeviceSynchronize();

    // Kopiowanie wynik�w z urz�dzenia na hosta
    hipMemcpy(isPrimeListHost, isPrimeListDevice, maxNumber * sizeof(bool), hipMemcpyDeviceToHost);

    // Czyszczenie pami�ci
    free(prepedPrimesHost);
    hipFree(isPrimeListDevice);
    hipFree(prepedPrimesDevice);
}
