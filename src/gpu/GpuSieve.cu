#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include "GpuSieve.hpp"
#include <iostream>
#include <vector>
#include <cassert>

// Maximum stride for GPU processing
const uint64_t MAX_STRIDE = 256000000;
// Number of threads per block (adjustable based on GPU)
int BLOCK_SIZE = 256;

// Constructor: Initialize GpuSieve object with no allocated memory on the host
GpuSieve::GpuSieve() : maxLimit_(0), isPrimeListHost_(nullptr) {}

// Destructor: Free host memory if allocated
GpuSieve::~GpuSieve() {}

// Set the maximum limit for prime computation
void GpuSieve::setMaxLimit(unsigned int maxLimit) {
    maxLimit_ = maxLimit;
}

// Main function to compute primes up to maxLimit
void GpuSieve::computePrimes() {
    // Prepare initial list of small primes on CPU using a simple sieve
    std::vector<uint64_t> prepedPrimes = sieveCpuPrep_(std::sqrt(maxLimit_));

    // Launch GPU sieve to find all primes up to maxLimit
    gpuSieve_(maxLimit_, prepedPrimes);
}

// Retrieve the list of primes found
const std::vector<unsigned int>& GpuSieve::getPrimes() const {
    return primes_;
}

// Collect primes from the result list on the host
void GpuSieve::collectPrimes() {
    // Clear any existing primes in the list
    primes_.clear();
    // Traverse isPrimeListHost to collect primes up to maxLimit
    for (uint64_t i = 2; i <= maxLimit_; i++) {
        if (isPrimeListHost_[i]) {
            primes_.push_back(i);
        }
    }
}

// Prepare a list of initial small primes on the CPU (sieve of Eratosthenes)
std::vector<uint64_t> GpuSieve::sieveCpuPrep_(uint64_t maxNumber) {
    std::vector<bool> isPrimeList(maxNumber, true);
    std::vector<uint64_t> prepedPrimes;

    // Basic sieve algorithm to mark non-primes
    for (uint64_t i = 2; i < maxNumber; i++) {
        if (isPrimeList[i]) {
            prepedPrimes.push_back(i);
            // Mark multiples of i as non-prime
            for (uint64_t j = 2; i * j < maxNumber; j++) {
                isPrimeList[i * j] = false;
            }
        }
    }
    return prepedPrimes;
}

// CUDA kernel to perform the sieve on the GPU
__global__ void gpuSieveKernel(uint64_t maxNumber, bool* isPrimeList, uint64_t* prepedPrimes, uint64_t sizeOfPrepedPrimes) {
    uint64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    // Iterate over each position in the range for this thread
    for (uint64_t i = threadIndex; i <= maxNumber; i += stride) {
        if (i < 2) continue;
        // Mark multiples of precomputed primes as non-prime
        for (uint64_t j = 0; j < sizeOfPrepedPrimes; j++) {
            uint64_t currentNumber = prepedPrimes[j] * i;
            if (currentNumber > maxNumber) break;
            isPrimeList[currentNumber] = false;
        }
    }
}

// Function to launch GPU sieve with prepared primes
void GpuSieve::gpuSieve_(uint64_t maxNumber, const std::vector<uint64_t>& prepedPrimes) {
    bool* isPrimeListDevice = nullptr;
    uint64_t* prepedPrimesDevice = nullptr;

    // Allocate and initialize memory on host and device
    isPrimeListHost_ = std::make_unique<bool[]>(maxNumber);
    std::memset(isPrimeListHost_.get(), true, maxNumber * sizeof(bool));
    hipMalloc(&isPrimeListDevice, maxNumber * sizeof(bool));
    hipMemcpy(isPrimeListDevice, isPrimeListHost_.get(), maxNumber * sizeof(bool), hipMemcpyHostToDevice);

    // Prepare and copy the list of initial primes to device
    uint64_t prepedPrimesSize = prepedPrimes.size();
    auto prepedPrimesHost = std::make_unique<uint64_t[]>(prepedPrimesSize);
    memcpy(prepedPrimesHost.get(), prepedPrimes.data(), prepedPrimesSize * sizeof(uint64_t));
    hipMalloc(&prepedPrimesDevice, prepedPrimesSize * sizeof(uint64_t));
    hipMemcpy(prepedPrimesDevice, prepedPrimesHost.get(), prepedPrimesSize * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Configure blocks and threads for kernel launch
    uint64_t numberOfBlocks = (maxNumber + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if (numberOfBlocks * BLOCK_SIZE > MAX_STRIDE) {
        numberOfBlocks = MAX_STRIDE / BLOCK_SIZE;
    }

    // Launch the kernel
    gpuSieveKernel << <numberOfBlocks, BLOCK_SIZE >> > (maxNumber, isPrimeListDevice, prepedPrimesDevice, prepedPrimesSize);
    hipDeviceSynchronize();

    // Copy results back to the host
    hipMemcpy(isPrimeListHost_.get(), isPrimeListDevice, maxNumber * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(isPrimeListDevice);
    hipFree(prepedPrimesDevice);
}
