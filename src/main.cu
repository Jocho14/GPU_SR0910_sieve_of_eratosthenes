﻿#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#include "iostream"
#include "memory"

#include "common/timer/Timer.hpp"

#include "cpu/singleThreaded/SingleCpuSieve.hpp"
#include "cpu/multiThreaded/MultiCpuSieve.hpp"
#include "gpu/GpuSieve.hpp"

int main() {
    auto timer = std::make_shared<Timer>();

    // Test dla SingleCpuSieve
    auto single = std::make_shared<SingleCpuSieve>(1000000);
    timer->start();
    single->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on single thread: " << timer->getTime() << "ms" << std::endl;

    timer->reset();

    // Test dla MultiCpuSieve
    auto multi = std::make_shared<MultiCpuSieve>(1000000);
    timer->start();
    multi->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on multi threads: " << timer->getTime() << "ms" << std::endl;

    timer->reset();

    // Test dla GpuSieve
    auto gpuSieve = std::make_shared<GpuSieve>();
    gpuSieve->setMaxLimit(1000000);
    timer->start();
    gpuSieve->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on GPU: " << timer->getTime() << "ms" << std::endl;

    // Zbieranie wyników
    single->collectPrimes();
    multi->collectPrimes();
    gpuSieve->collectPrimes();

    // Wyświetlanie liczby liczb pierwszych
    std::cout << "NUM OF PRIMES FOR SINGLE: " << single->getPrimes().size() << std::endl;
    std::cout << "NUM OF PRIMES FOR MULTI: " << multi->getPrimes().size() << std::endl;
    std::cout << "NUM OF PRIMES FOR GPU: " << gpuSieve->getPrimes().size() << std::endl;

    // Porównanie wyników między implementacjami
    if (single->getPrimes().size() == multi->getPrimes().size() &&
        multi->getPrimes().size() == gpuSieve->getPrimes().size()) {
        std::cout << "All implementations produced the same number of primes." << std::endl;
    }
    else {
        std::cout << "Mismatch in the number of primes found by different implementations." << std::endl;
    }

    return 0;
}