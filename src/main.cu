﻿#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include "memory"
#include <string>

#include "common/timer/Timer.hpp"

#include "cpu/singleThreaded/SingleCpuSieve.hpp"
#include "cpu/multiThreaded/MultiCpuSieve.hpp"
#include "gpu/GpuSieve.hpp"
#include "../src/test/Test.hpp"

int main() {

	auto single = std::make_shared<SingleCpuSieve>(42949);
	auto multi = std::make_shared<MultiCpuSieve>(42949);
	auto gpu = std::make_shared<GpuSieve>();
	/*auto gpuNcite = std::make_shared<>();*/

	std::ofstream outFileSingleTime("singleTime.csv");
	std::ofstream outFileMultiTime("multiTime.csv");
	std::ofstream outFileGpuTime("GpuTime.csv");
	std::ofstream outFileGpuTimeNcite("GpuTimeNcite.csv");

	std::ofstream outFileSingleUsage("singleUsage.csv");
	std::ofstream outFileMultiUsage("multiUsage.csv");
	std::ofstream outFileGpuUsage("GpuUsage.csv");
	std::ofstream outFileGpuUsageNcite("GpuUsageNcite.csv");

	/*const unsigned int Max = 4294967285;*/
	const unsigned int Max = 42949;

	auto test = std::make_shared<Test>();

	std::cout << "start time test for singlecpusieve...\n";
	test->runTimeTest(Max, single, outFileSingleTime);
	std::cout << "stop time test for singlecpusieve.\n";
	
	std::cout << "start time test for multicpusieve...\n";
	test->runTimeTest(Max, multi, outFileMultiTime);
	std::cout << "stop time test for multicpusieve.\n";

	std::cout << "start time test for gpusieve...\n";
	test->runTimeTest(Max, gpu, outFileGpuTime);
	std::cout << "stop time test for gpusieve.\n";

	/*std::cout << "start time test for gpusieve after ncite...\n";
	test->runTimeTest(Max, gpuNcite, outFileGpuTimeNcite);
	std::cout << "stop time test for gpusieve after ncite.\n";*/

	std::cout << "start usage test for singlecpusieve...\n";
	test->runUsageTest(Max, single, outFileSingleUsage, false);
	std::cout << "stop usage test for singlecpusieve.\n";

	std::cout << "start usage test for multicpusieve...\n";
	test->runUsageTest(Max, multi, outFileMultiUsage, false);
	std::cout << "stop usage test for multicpusieve.\n";

	//std::cout << "Start usage test for GpuSieve...\n";
	//test->runUsageTest(Max, gpu, outFileGpuUsage, true);
	//std::cout << "Stop usage test for GpuSieve.\n";

	/*std::cout << "start time test for gpusieve after ncite...\n";
	test->runTimeTest(Max, gpuNcite, outFileGpuUsageNcite, true);
	std::cout << "stop time test for gpusieve after ncite.\n";*/

	return 0;
}