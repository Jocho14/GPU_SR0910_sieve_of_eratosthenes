﻿#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#include "iostream"
#include "memory"

#include "common/timer/Timer.hpp"

#include "cpu/singleThreaded/SingleCpuSieve.hpp"
#include "cpu/multiThreaded/MultiCpuSieve.hpp"
#include "gpu/GpuSieve.hpp"

int main() {
    auto timer = std::make_shared<Timer>();

    // Test for SingleCpuSieve
    auto single = std::make_shared<SingleCpuSieve>(1000000);
    timer->start();
    single->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on single thread: " << timer->getTime() << "ms" << std::endl;

    timer->reset();

    // Test for MultiCpuSieve
    auto multi = std::make_shared<MultiCpuSieve>(1000000);
    timer->start();
    multi->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on multi threads: " << timer->getTime() << "ms" << std::endl;

    timer->reset();

    // Test for GpuSieve
    auto gpuSieve = std::make_shared<GpuSieve>();
    gpuSieve->setMaxLimit(1000000);
    timer->start();
    gpuSieve->computePrimes();
    timer->stop();
    std::cout << "Time taken to compute primes on GPU: " << timer->getTime() << "ms" << std::endl;

    // results
    single->collectPrimes();
    multi->collectPrimes();
    gpuSieve->collectPrimes();

    
    std::cout << "NUM OF PRIMES FOR SINGLE: " << single->getPrimes().size() << std::endl;
    std::cout << "NUM OF PRIMES FOR MULTI: " << multi->getPrimes().size() << std::endl;
    std::cout << "NUM OF PRIMES FOR GPU: " << gpuSieve->getPrimes().size() << std::endl;

    // Comparing results
    if (single->getPrimes().size() == multi->getPrimes().size() &&
        multi->getPrimes().size() == gpuSieve->getPrimes().size()) {
        std::cout << "All implementations produced the same number of primes." << std::endl;
    }
    else {
        std::cout << "Mismatch in the number of primes found by different implementations." << std::endl;
    }

    return 0;
}